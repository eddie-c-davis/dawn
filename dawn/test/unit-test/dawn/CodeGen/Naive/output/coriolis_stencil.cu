#include "hip/hip_runtime.h"
// gtclang (0.0.4-5fc8d35e-x86_64-GNU-9.2.1)
// based on LLVM/Clang (9.0.0), Dawn (0.0.2)
// Generated on 2020-02-20  14:11:44

#define DAWN_GENERATED 1
#define DAWN_BACKEND_T CUDA
#ifndef BOOST_RESULT_OF_USE_TR1
 #define BOOST_RESULT_OF_USE_TR1 1
#endif
#ifndef BOOST_NO_CXX11_DECLTYPE
 #define BOOST_NO_CXX11_DECLTYPE 1
#endif
#ifndef GRIDTOOLS_DAWN_HALO_EXTENT
 #define GRIDTOOLS_DAWN_HALO_EXTENT 3
#endif
#ifndef BOOST_PP_VARIADICS
 #define BOOST_PP_VARIADICS 1
#endif
#ifndef BOOST_FUSION_DONT_USE_PREPROCESSED_FILES
 #define BOOST_FUSION_DONT_USE_PREPROCESSED_FILES 1
#endif
#ifndef BOOST_MPL_CFG_NO_PREPROCESSED_HEADERS
 #define BOOST_MPL_CFG_NO_PREPROCESSED_HEADERS 1
#endif
#ifndef GT_VECTOR_LIMIT_SIZE
 #define GT_VECTOR_LIMIT_SIZE 30
#endif
#ifndef BOOST_FUSION_INVOKE_MAX_ARITY
 #define BOOST_FUSION_INVOKE_MAX_ARITY GT_VECTOR_LIMIT_SIZE
#endif
#ifndef FUSION_MAX_VECTOR_SIZE
 #define FUSION_MAX_VECTOR_SIZE GT_VECTOR_LIMIT_SIZE
#endif
#ifndef FUSION_MAX_MAP_SIZE
 #define FUSION_MAX_MAP_SIZE GT_VECTOR_LIMIT_SIZE
#endif
#ifndef BOOST_MPL_LIMIT_VECTOR_SIZE
 #define BOOST_MPL_LIMIT_VECTOR_SIZE GT_VECTOR_LIMIT_SIZE
#endif
#include <driver-includes/gridtools_includes.hpp>
using namespace gridtools::dawn;
//===--------------------------------------------------------------------------------*- C++ -*-===//
//                         _       _
//                        | |     | |
//                    __ _| |_ ___| | __ _ _ __   __ _
//                   / _` | __/ __| |/ _` | '_ \ / _` |
//                  | (_| | || (__| | (_| | | | | (_| |
//                   \__, |\__\___|_|\__,_|_| |_|\__, | - GridTools Clang DSL
//                    __/ |                       __/ |
//                   |___/                       |___/
//
//
//  This file is distributed under the MIT License (MIT).
//  See LICENSE.txt for details.
//
//===------------------------------------------------------------------------------------------===//

#include "gtclang_dsl_defs/gtclang_dsl.hpp"

using namespace gtclang::dsl;

namespace dawn_generated {
namespace cuda {
__global__ void __launch_bounds__(128)
    coriolis_stencil_stencil50_ms104_kernel(const int isize, const int jsize, const int ksize, const int stride_111_1,
                                            const int stride_111_2, ::dawn::float_type* const u_nnow,
                                            ::dawn::float_type* const v_nnow, ::dawn::float_type* const fc,
                                            ::dawn::float_type* const u_tens, ::dawn::float_type* const v_tens) {
  // Start kernel
  const unsigned int nx = isize;
  const unsigned int ny = jsize;
  const int block_size_i = (blockIdx.x + 1) * 32 < nx ? 32 : nx - blockIdx.x * 32;
  const int block_size_j = (blockIdx.y + 1) * 4 < ny ? 4 : ny - blockIdx.y * 4;

  // computing the global position in the physical domain

  // In a typical cuda block we have the following regions

  // aa bbbbbbbb cc

  // aa bbbbbbbb cc

  // hh dddddddd ii

  // hh dddddddd ii

  // hh dddddddd ii

  // hh dddddddd ii

  // ee ffffffff gg

  // ee ffffffff gg

  // Regions b,d,f have warp (or multiple of warp size)

  // Size of regions a, c, h, i, e, g are determined by max_extent_t

  // Regions b,d,f are easily executed by dedicated warps (one warp for each line)

  // Regions (a,h,e) and (c,i,g) are executed by two specialized warp
  int iblock = 0 - 1;
  int jblock = 0 - 1;
  if (threadIdx.y < +4) {
    iblock = threadIdx.x;
    jblock = (int)threadIdx.y + 0;
  }
  // initialized iterators
  int idx111 = (blockIdx.x * 32 + iblock) * 1 + (blockIdx.y * 4 + jblock) * stride_111_1;

  // jump iterators to match the intersection of beginning of next interval and the parallel execution block
  idx111 += max(0, blockIdx.z * 4) * stride_111_2;
  int kleg_lower_bound = max(0, blockIdx.z * 4);
  int kleg_upper_bound = min(ksize - 1 + 0, (blockIdx.z + 1) * 4 - 1);
  ;
  for (int k = kleg_lower_bound + 0; k <= kleg_upper_bound + 0; ++k) {
    if (iblock >= 0 && iblock <= block_size_i - 1 + 0 && jblock >= 0 && jblock <= block_size_j - 1 + 0) {
      ::dawn::float_type __local_z_fv_north_98 =
          (__ldg(&(fc[idx111])) * (__ldg(&(v_nnow[idx111])) + __ldg(&(v_nnow[idx111 + 1 * 1]))));
      ::dawn::float_type __local_z_fv_south_99 =
          (__ldg(&(fc[idx111 + stride_111_1 * -1])) *
           (__ldg(&(v_nnow[idx111 + stride_111_1 * -1])) + __ldg(&(v_nnow[idx111 + 1 * 1 + stride_111_1 * -1]))));
      u_tens[idx111] += ((::dawn::float_type)0.25 * (__local_z_fv_north_98 + __local_z_fv_south_99));
      ::dawn::float_type __local_z_fu_east_101 =
          (__ldg(&(fc[idx111])) * (__ldg(&(u_nnow[idx111])) + __ldg(&(u_nnow[idx111 + stride_111_1 * 1]))));
      ::dawn::float_type __local_z_fu_west_102 =
          (__ldg(&(fc[idx111 + 1 * -1])) *
           (__ldg(&(u_nnow[idx111 + 1 * -1])) + __ldg(&(u_nnow[idx111 + 1 * -1 + stride_111_1 * 1]))));
      v_tens[idx111] -= ((::dawn::float_type)0.25 * (__local_z_fu_east_101 + __local_z_fu_west_102));
    }
    // Slide kcaches

    // increment iterators
    idx111 += stride_111_2;
  }
}

class coriolis_stencil {
 public:
  struct sbase : public timer_cuda {
    sbase(std::string name) : timer_cuda(name) {}

    double get_time() { return total_time(); }
  };

  struct stencil_50 : public sbase {
    // Members

    // Temporary storage typedefs
    using tmp_halo_t = gridtools::halo<0, 0, 0, 0, 0>;
    using tmp_meta_data_t = storage_traits_t::storage_info_t<0, 5, tmp_halo_t>;
    using tmp_storage_t = storage_traits_t::data_store_t<::dawn::float_type, tmp_meta_data_t>;
    const gridtools::dawn::domain m_dom;

   public:
    stencil_50(const gridtools::dawn::domain& dom_, int rank, int xcols, int ycols)
        : sbase("stencil_50"), m_dom(dom_) {}

    void run(storage_ijk_t u_nnow_ds, storage_ijk_t v_nnow_ds, storage_ijk_t fc_ds,
             storage_ijk_t u_tens_ds, storage_ijk_t v_tens_ds) {
      // starting timers
      start();
      {
        ;
        gridtools::data_view<storage_ijk_t> u_nnow = gridtools::make_device_view(u_nnow_ds);
        gridtools::data_view<storage_ijk_t> v_nnow = gridtools::make_device_view(v_nnow_ds);
        gridtools::data_view<storage_ijk_t> fc = gridtools::make_device_view(fc_ds);
        gridtools::data_view<storage_ijk_t> u_tens = gridtools::make_device_view(u_tens_ds);
        gridtools::data_view<storage_ijk_t> v_tens = gridtools::make_device_view(v_tens_ds);
        const unsigned int nx = m_dom.isize() - m_dom.iminus() - m_dom.iplus();
        const unsigned int ny = m_dom.jsize() - m_dom.jminus() - m_dom.jplus();
        const unsigned int nz = m_dom.ksize() - m_dom.kminus() - m_dom.kplus();
        dim3 threads(32, 4 + 0, 1);
        const unsigned int nbx = (nx + 32 - 1) / 32;
        const unsigned int nby = (ny + 4 - 1) / 4;
        const unsigned int nbz = (m_dom.ksize() + 4 - 1) / 4;
        dim3 blocks(nbx, nby, nbz);
        coriolis_stencil_stencil50_ms104_kernel<<<blocks, threads>>>(
            nx, ny, nz, u_tens_ds.strides()[1], u_tens_ds.strides()[2],
            (u_nnow.data() + u_nnow_ds.get_storage_info_ptr()->index(u_nnow.begin<0>(), u_nnow.begin<1>(), 0)),
            (v_nnow.data() + v_nnow_ds.get_storage_info_ptr()->index(v_nnow.begin<0>(), v_nnow.begin<1>(), 0)),
            (fc.data() + fc_ds.get_storage_info_ptr()->index(fc.begin<0>(), fc.begin<1>(), 0)),
            (u_tens.data() + u_tens_ds.get_storage_info_ptr()->index(u_tens.begin<0>(), u_tens.begin<1>(), 0)),
            (v_tens.data() + v_tens_ds.get_storage_info_ptr()->index(v_tens.begin<0>(), v_tens.begin<1>(), 0)));
      };

      // stopping timers
      pause();
    }
  };
  static constexpr const char* s_name = "coriolis_stencil";
  stencil_50 m_stencil_50;

 public:
  coriolis_stencil(const coriolis_stencil&) = delete;

  // Members

  // Stencil-Data

  coriolis_stencil(const gridtools::dawn::domain& dom, int rank = 1, int xcols = 1, int ycols = 1)
      : m_stencil_50(dom, rank, xcols, ycols) {}

  template <typename S>
  void sync_storages(S field) {
    field.sync();
  }

  template <typename S0, typename... S>
  void sync_storages(S0 f0, S... fields) {
    f0.sync();
    sync_storages(fields...);
  }

  void run(storage_ijk_t u_nnow, storage_ijk_t v_nnow, storage_ijk_t fc, storage_ijk_t u_tens, storage_ijk_t v_tens) {
    sync_storages(u_nnow, v_nnow, fc, u_tens, v_tens);
    m_stencil_50.run(u_nnow, v_nnow, fc, u_tens, v_tens);
    sync_storages(u_nnow, v_nnow, fc, u_tens, v_tens);
  }

  std::string get_name() const { return std::string(s_name); }

  void reset_meters() { m_stencil_50.reset(); }

  double get_total_time() {
    double res = 0;
    res += m_stencil_50.get_time();
    return res;
  }
};
}  // namespace cuda
}  // namespace dawn_generated

#include "driver-includes/verify.hpp"
#include <iostream>
#include <iomanip>
#include <cstdlib>

void print(const domain& dom, const gridtools::data_view<storage_ijk_t>& view) {
  for(int i = dom.iminus(); i < std::min(int(dom.isize() - dom.iplus()), view.total_length<0>()); ++i)
    for(int j = dom.jminus(); j < std::min(int(dom.jsize() - dom.jplus()), view.total_length<1>()); ++j)
      for(int k = dom.kminus(); k < std::min(int(dom.ksize() - dom.kplus()), view.total_length<2>()); ++k)
        std::cout << std::setprecision(9) << view(i, j, k) << ' ';
  std::cout << std::endl;
}

int main(int argc, const char** argv) {
  int isize, jsize, ksize, halo;
  if(argc > 1)
    isize = jsize = ksize = atoi(argv[1]);
  else
    isize = jsize = ksize = 12;
  if(argc > 2)
    halo = atoi(argv[2]);
  else
    halo = GRIDTOOLS_DAWN_HALO_EXTENT;

  float time;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  domain dom(isize, jsize, ksize);
  dom.set_halos(halo, halo, halo, halo, 0, 0);
  meta_data_t meta(isize, jsize, ksize+1);
  storage_t u_nnow(meta, "u_nnow"),v_nnow(meta, "v_nnow"),fc(meta, "fc"),u_tens(meta, "u_tens"),v_tens(meta, "v_tens");
  verifier verif(dom);
  verif.fillMath(8,2,1.5,1.5,2,4,u_nnow);
  verif.fillMath(5,1.2,1.3,1.7,2.2,3.5,v_nnow);
  verif.fillMath(2,1.3,1.4,1.6,2.1,3,fc);
  verif.fill(-1,u_tens,v_tens);

  dawn_generated::cuda::coriolis_stencil stencil(dom);
  hipEventRecord(start, 0);
  stencil.run(u_nnow,v_nnow,fc,u_tens,v_tens);
  hipEventRecord(stop, 0);

  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  std::cerr << "cuda_time = " << (time * 1E-3)  << std::endl;

  //print(dom, make_host_view(u_tens));
  //print(dom, make_host_view(v_tens));
}
